/*$Id: map.cu 727 2009-11-11 11:32:44Z wenbinor $*/
/**
 *This is the source code for Mars, a MapReduce framework on graphics
 *processors.
 *Developers: Wenbin Fang (HKUST), Bingsheng He (Microsoft Research Asia)
 *Naga K. Govindaraju (Microsoft Corp.), Qiong Luo (HKUST), Tuyong Wang (Sina.com).
 *If you have any question on the code, please contact us at 
 *           wenbin@cse.ust.hk or savenhe@microsoft.com
 *
 *The license is a free non-exclusive, non-transferable license to reproduce, 
 *use, modify and display the source code version of the Software, with or 
 *without modifications solely for non-commercial research, educational or 
 *evaluation purposes. The license does not entitle Licensee to technical support, 
 *telephone assistance, enhancements or updates to the Software. All rights, title 
 *to and ownership interest in Mars, including all intellectual property rights 
 *therein shall remain in HKUST.
 */

#ifndef __MAP_CU__
#define __MAP_CU__

#include "MarsInc.h"
#include "global.h"

__device__ void MAP_COUNT_FUNC//(void *key, void *val, size_t keySize, size_t valSize)
{
	EMIT_INTER_COUNT_FUNC(sizeof(int),sizeof(VAL));
}

__device__ void MAP_FUNC//(void *key, void *val, size_t keySize, size_t valSize)
{
	int* pKey = (int*)key;
	VAL* pVal = (VAL*)val;
	//这一部分必须要加上，否则内存会报错

	//VAL* val_lable = (VAL*)GET_OUTPUT_BUF(sizeof(VAL));
VAL* val_lable = (VAL*)GET_OUTPUT_BUF(sizeof(0));
	//int* o_key = (int*)GET_OUTPUT_BUF(sizeof(int));

	int o_key=*pKey;
	val_lable->label=pVal->label;
	val_lable->value=pVal->value;
	EMIT_INTERMEDIATE_FUNC( &o_key,val_lable,sizeof(int),sizeof(VAL));
}
#endif //__MAP_CU__
